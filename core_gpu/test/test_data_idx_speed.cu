#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <iostream>
#include <cstring>
#include <chrono>
#include <limits>

#include <smmintrin.h>

#include "octnet/gpu/gpu.h"
#include "octnet/gpu/common.h"
#include "octnet/cpu/cpu.h"
#include "octnet/test/objects.h"



// #ifdef __CUDA_ARCH__
// __host__ __device__ 
// #endif
// OCTREE_FUNCTION
// inline int tree_data_idx2(const ot_tree_t* tree, int bit_idx, ot_size_t feature_size) {
//   // int pa_idx = IMAX(0, tree_parent_bit_idx(bit_idx));
//   // if(!tree_isset_bit(tree, pa_idx)) {
//   //   bit_idx = pa_idx;
//   //   pa_idx = tree_parent_bit_idx(pa_idx);
//   // }
//   // if(!tree_isset_bit(tree, pa_idx)) {
//   //   bit_idx = pa_idx;
//   //   pa_idx = tree_parent_bit_idx(pa_idx);
//   // }
//   // if(!tree_isset_bit(tree, pa_idx)) {
//   //   return 0;
//   // }

//   int pa_idx = tree_parent_bit_idx(bit_idx);
//   int papa_idx = tree_parent_bit_idx(pa_idx);
//   int papapa_idx = tree_parent_bit_idx(papa_idx);
//   bit_idx = IMAX(bit_idx * tree_isset_bit(tree, pa_idx), IMAX(pa_idx * tree_isset_bit(tree, papa_idx), papa_idx * tree_isset_bit(tree, papapa_idx)));
//   if(bit_idx == 0) {
//     return 0;
//   }
//   pa_idx = IMAX(0, tree_parent_bit_idx(bit_idx));


//   // int data_idx = tree_cnt0(tree, 0, IMIN(bit_idx, 73)); 
//   // if(pa_idx > 1) {
//   //   data_idx -= 8 * tree_cnt0(tree, 1, pa_idx);
//   // }
//   // if(bit_idx > 72) {
//   //   data_idx += bit_idx - 73;
//   // }
  
//   int data_idx = tree_cnt1(tree, 0, pa_idx);
//   data_idx = data_idx * 8 + 1 
//              + (bit_idx-1)%8 
//              - (data_idx + tree_cnt1(tree, pa_idx, bit_idx));

//   return data_idx * feature_size;
// }




// void correctness_cpu(ot_tree_t* tree) {
//   std::cout << "---------------------- test correctness cpu --------------------" << std::endl;
//   std::cout << tree_bit_str(tree) << std::endl;
//   for(int bit_idx = 0; bit_idx < (1+8+64+64*8); ++bit_idx) {
//     int di_gt = tree_data_idx(tree, bit_idx, 1);
//     int di2 = tree_data_idx2(tree, bit_idx, 1);

//     if(!tree_isset_bit(tree, bit_idx) && di2 != di_gt) {
//       std::cout << "[ERROR_CPU] bit_idx=" << bit_idx << ": " << di2 << " should be " << di_gt << std::endl;
//     }
//   }
// }

// void speed_cpu(ot_tree_t* tree) {
//   int reps = 100000;

//   int di = 0;
//   auto t1 = std::chrono::high_resolution_clock::now();
//   for(int rep = 0; rep < reps; ++rep) {
//     for(int bit_idx = 0; bit_idx < (1+8+64+64*8); ++bit_idx) {
//       int tmp = tree_data_idx(tree, bit_idx, 1);
//       di += tmp;
//     }
//   }
//   auto t2 = std::chrono::high_resolution_clock::now();
//   auto time_span = std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1);
//   std::cout << "cpu old took " << time_span.count() << "[s]" << std::endl;
//   std::cout << di << std::endl;

//   di = 0;
//   t1 = std::chrono::high_resolution_clock::now();
//   for(int rep = 0; rep < reps; ++rep) {
//     for(int bit_idx = 0; bit_idx < (1+8+64+64*8); ++bit_idx) {
//       int tmp = tree_data_idx2(tree, bit_idx, 1);
//       di += tmp;
//     }
//   }
//   t2 = std::chrono::high_resolution_clock::now();
//   time_span = std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1);
//   std::cout << "cpu new took " << time_span.count() << "[s]" << std::endl;
//   std::cout << di << std::endl;
// }





// __global__ void kernel_correctness(const ot_tree_t* tree, int n_bit_ind) {
//   CUDA_KERNEL_LOOP(bit_idx, n_bit_ind) { 
//     int di_gt = tree_data_idx(tree, bit_idx, 1);
//     int di2 = tree_data_idx2(tree, bit_idx, 1);
//     if(!tree_isset_bit(tree, bit_idx) && di2 != di_gt) {
//       printf("[ERROR_GPU] bit_idx=%d: %d should be %d\n", bit_idx, di2, di_gt);
//     }
//   }
// }

// void correctness_gpu(ot_tree_t* tree_h) {
//   std::cout << "---------------------- test correctness gpu --------------------" << std::endl;
//   std::cout << tree_bit_str(tree_h) << std::endl;
  
//   ot_tree_t* tree_d = host_to_device_malloc(tree_h, N_TREE_INTS);

//   int n_bit_ind = 1+8+64+64*8;
//   kernel_correctness<<<GET_BLOCKS(n_bit_ind), CUDA_NUM_THREADS>>>(
//       tree_d, n_bit_ind 
//   );
//   CUDA_POST_KERNEL_CHECK;

//   device_free(tree_d);
// }

// __global__ void kernel_speed1(const ot_tree_t* tree, int n_bit_ind) {
//   CUDA_KERNEL_LOOP(bit_idx, n_bit_ind) { 
//     int di = tree_data_idx(tree, bit_idx, 1);
//     if(di > 1000000 ) {
//       printf("[ERROR_GPU] you summoned an evil demon\n");
//     }
//   }
// }
// __global__ void kernel_speed2(const ot_tree_t* tree, int n_bit_ind) {
//   CUDA_KERNEL_LOOP(bit_idx, n_bit_ind) { 
//     int di = tree_data_idx2(tree, bit_idx, 1);
//     if(di > 1000000 ) {
//       printf("[ERROR_GPU] you summoned an evil demon\n");
//     }
//   }
// }

// void speed_gpu(ot_tree_t* tree_h) {
//   int reps = 100000;
//   int n_bit_ind = 1+8+64+64*8;
  
//   ot_tree_t* tree_d = host_to_device_malloc(tree_h, N_TREE_INTS);

//   kernel_speed1<<<GET_BLOCKS(n_bit_ind), CUDA_NUM_THREADS>>>(tree_d, n_bit_ind);
//   auto t1 = std::chrono::high_resolution_clock::now();
//   for(int rep = 0; rep < reps; ++rep) {
//     kernel_speed1<<<GET_BLOCKS(n_bit_ind), CUDA_NUM_THREADS>>>(tree_d, n_bit_ind);
//     CUDA_POST_KERNEL_CHECK;
//   }
//   auto t2 = std::chrono::high_resolution_clock::now();
//   auto time_span = std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1);
//   std::cout << "gpu old took " << time_span.count() << "[s]" << std::endl;

//   kernel_speed2<<<GET_BLOCKS(n_bit_ind), CUDA_NUM_THREADS>>>(tree_d, n_bit_ind);
//   t1 = std::chrono::high_resolution_clock::now();
//   for(int rep = 0; rep < reps; ++rep) {
//     kernel_speed2<<<GET_BLOCKS(n_bit_ind), CUDA_NUM_THREADS>>>(tree_d, n_bit_ind);
//     CUDA_POST_KERNEL_CHECK;
//   }
//   t2 = std::chrono::high_resolution_clock::now();
//   time_span = std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1);
//   std::cout << "gpu new took " << time_span.count() << "[s]" << std::endl;
  
//   device_free(tree_d);
// }



int main(int argc, char** argv) {
//   std::cout << "[IMPROVE] tree_data_idx" << std::endl;

//   ot_tree_t* tree = new ot_tree_t[N_TREE_INTS];

//   //test set 1
//   memset(tree, 0, N_TREE_INTS * sizeof(ot_tree_t));
//   correctness_cpu(tree);
//   correctness_gpu(tree);
//   speed_cpu(tree);
//   speed_gpu(tree);

//   //test set 1
//   memset(tree, 0, N_TREE_INTS * sizeof(ot_tree_t));
//   tree_set_bit(tree, 0);
//   tree_set_bit(tree, 1);
//   tree_set_bit(tree, 2);
//   tree_set_bit(tree, 9);
//   tree_set_bit(tree, 10);
//   tree_set_bit(tree, 18);
//   correctness_cpu(tree);
//   correctness_gpu(tree);
//   speed_cpu(tree);
//   speed_gpu(tree);

//   //test set 2
//   memset(tree, 0, N_TREE_INTS * sizeof(ot_tree_t));
//   tree_set_bit(tree, 0);
//   tree_set_bit(tree, 1);
//   tree_set_bit(tree, 9);
//   tree_set_bit(tree, 10);
//   tree_set_bit(tree, 11);
//   tree_set_bit(tree, 4);
//   tree_set_bit(tree, 5);
//   tree_set_bit(tree, 8);
//   tree_set_bit(tree, 65);
//   tree_set_bit(tree, 66);
//   tree_set_bit(tree, 72);
//   correctness_cpu(tree);
//   correctness_gpu(tree);
//   speed_cpu(tree);
//   speed_gpu(tree);

//   //test set 3
//   memset(tree, 0, N_TREE_INTS * sizeof(ot_tree_t));
//   for(int idx = 0; idx < 73; ++idx) { tree_set_bit(tree, idx); }
//   correctness_cpu(tree);
//   correctness_gpu(tree);
//   speed_cpu(tree);
//   speed_gpu(tree);

//   delete[] tree;
//   std::cout << "[DONE]" << std::endl;
  
  return 0;
}
