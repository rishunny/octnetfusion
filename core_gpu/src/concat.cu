#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "octnet/gpu/combine.h"
#include "octnet/gpu/gpu.h"

#include <cstdio>
#include <cstdlib>



__global__ void kernel_concat(ot_data_t* out, int n_leafs, const ot_data_t* in1, const ot_data_t* in2, const ot_size_t feature_size_in1, const ot_size_t feature_size_in2, const ot_size_t feature_size_out) {
  CUDA_KERNEL_LOOP(vx_idx, n_leafs) {
    octree_cpy_leaf(in1 + vx_idx * feature_size_in1, feature_size_in1, out + vx_idx * feature_size_out);
    octree_cpy_leaf(in2 + vx_idx * feature_size_in2, feature_size_in2, out + vx_idx * feature_size_out + feature_size_in1);
  }
}

void octree_concat_gpu(const octree* in1, const octree* in2, bool check, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_concat_gpu\n"); }

  if(check && (!octree_equal_trees_gpu(in1, in2))) {
    printf("ERROR: tree structure of inputs do not match\n");
    exit(-1);
  }

  ot_size_t feature_size_in1 = in1->feature_size;
  ot_size_t feature_size_in2 = in2->feature_size;
  ot_size_t feature_size_out = feature_size_in1 + feature_size_in2;

  octree_resize_gpu(in1->n, in1->grid_depth, in1->grid_height, in1->grid_width, feature_size_out, in1->n_leafs, out);
  octree_cpy_trees_gpu_gpu(in1, out);
  octree_cpy_prefix_leafs_gpu_gpu(in1, out);

  kernel_concat<<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
      out->data, in1->n_leafs, in1->data, in2->data, feature_size_in1, feature_size_in2, feature_size_out
  );
  CUDA_POST_KERNEL_CHECK;
}


template <bool do_grad_in2>
__global__ void kernel_concat_bwd(ot_data_t* grad_in1, ot_data_t* grad_in2, int n_leafs, const ot_data_t* grad_out, const ot_size_t feature_size_in1, const ot_size_t feature_size_in2, const ot_size_t feature_size_out) {
  CUDA_KERNEL_LOOP(vx_idx, n_leafs) {
    octree_cpy_leaf(grad_out + vx_idx * feature_size_out, feature_size_in1, grad_in1 + vx_idx * feature_size_in1);
    if(do_grad_in2) {
      octree_cpy_leaf(grad_out + vx_idx * feature_size_out + feature_size_in1, feature_size_in2, grad_in2 + vx_idx * feature_size_in2);
    }
  }
}

void octree_concat_bwd_gpu(const octree* in1, const octree* in2, const octree* grad_out, bool do_grad_in2, octree* grad_in1, octree* grad_in2) {
  if(DEBUG) { printf("[DEBUG] octree_concat_bwd_gpu\n"); }

  octree_resize_as_gpu(in1, grad_in1);
  octree_cpy_trees_gpu_gpu(in1, grad_in1);
  octree_cpy_prefix_leafs_gpu_gpu(in1, grad_in1);

  ot_size_t feature_size_in1 = in1->feature_size;
  ot_size_t feature_size_in2 = in2->feature_size;
  ot_size_t feature_size_out = feature_size_in1 + feature_size_in2;

  if(do_grad_in2) {
    octree_resize_as_gpu(in2, grad_in2);
    octree_cpy_trees_gpu_gpu(in2, grad_in2);
    octree_cpy_prefix_leafs_gpu_gpu(in2, grad_in2);

    kernel_concat_bwd<true><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2->data, in1->n_leafs, grad_out->data, feature_size_in1, feature_size_in2, feature_size_out
    );
  }
  else {
    kernel_concat_bwd<false><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2->data, in1->n_leafs, grad_out->data, feature_size_in1, feature_size_in2, feature_size_out
    );

  }
  CUDA_POST_KERNEL_CHECK;
}



__global__ void kernel_concat_ds(octree out, int n_leafs, const octree in1, const octree in2) {
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    int out_idx = leaf_idx * out.feature_size;
    octree_cpy_leaf(in1.data + leaf_idx * in1.feature_size, in1.feature_size, out.data + out_idx);
    
    const int in1_grid_idx = leaf_idx_to_grid_idx(&in1, leaf_idx); 
    const ot_tree_t* in1_tree = octree_get_tree(&in1, in1_grid_idx);

    int in1_data_idx = leaf_idx - in1.prefix_leafs[in1_grid_idx];
    int in1_bit_idx = data_idx_to_bit_idx(in1_tree, in1_data_idx);

    int n,ds,hs,ws;
    int cell_depth = octree_ind_to_dense_ind(&in1, in1_grid_idx, in1_bit_idx, &n, &ds,&hs,&ws);
    int cell_width = width_from_depth(cell_depth);

    for(int f = 0; f < in2.feature_size; ++f) {
      out.data[out_idx + in1.feature_size + f] = 0;
    }

    for(int d = ds; d < (ds+cell_width); ++d) {
      for(int h = hs; h < (hs+cell_width); ++h) {
        for(int w = ws; w < (ws+cell_width); ++w) {
          int gd = d / 8;
          int gh = h / 8;
          int gw = w / 8;
          int bd = d % 8;
          int bh = h % 8;
          int bw = w % 8;

          int in2_grid_idx = octree_grid_idx(&in2, n, gd,gh,gw);
          const ot_tree_t* in2_tree = octree_get_tree(&in2, in2_grid_idx);
          int in2_bit_idx = tree_bit_idx(in2_tree, bd,bh,bw);
          const ot_data_t* in2_data = octree_get_data(&in2, in2_grid_idx);
          int in2_data_idx = tree_data_idx(in2_tree, in2_bit_idx, in2.feature_size);
          
          for(int f = 0; f < in2.feature_size; ++f) {
            out.data[out_idx + in1.feature_size + f] += in2_data[in2_data_idx + f];
          }
        }
      }
    }
    
    for(int f = 0; f < in2.feature_size; ++f) {
      out.data[out_idx + in1.feature_size + f] /= (cell_width*cell_width*cell_width);
    }
  }
}

void octree_concat_ds_gpu(const octree* in1, const octree* in2, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_concat_ds_gpu\n"); }

  ot_size_t feature_size_in1 = in1->feature_size;
  ot_size_t feature_size_in2 = in2->feature_size;
  ot_size_t feature_size_out = feature_size_in1 + feature_size_in2;

  octree_resize_gpu(in1->n, in1->grid_depth, in1->grid_height, in1->grid_width, feature_size_out, in1->n_leafs, out);
  octree_cpy_trees_gpu_gpu(in1, out);
  octree_cpy_prefix_leafs_gpu_gpu(in1, out);

  kernel_concat_ds<<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
      *out, in1->n_leafs, *in1, *in2
  );
  CUDA_POST_KERNEL_CHECK;
}


__global__ void kernel_concat_ds_bwd1(octree grad_in1, int n_leafs, const octree grad_out) {
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    int out_idx = leaf_idx * grad_out.feature_size;
    octree_cpy_leaf(grad_out.data + out_idx, grad_in1.feature_size, grad_in1.data + leaf_idx * grad_in1.feature_size);
  }
}

__global__ void kernel_concat_ds_bwd2(octree grad_in2, int n_leafs, const octree grad_out, int in1_feature_size) {
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    const int in2_grid_idx = leaf_idx_to_grid_idx(&grad_in2, leaf_idx);   
    const ot_tree_t* in2_tree = octree_get_tree(&grad_in2, in2_grid_idx);

    int in2_data_idx = leaf_idx - grad_in2.prefix_leafs[in2_grid_idx];
    int in2_bit_idx = data_idx_to_bit_idx(in2_tree, in2_data_idx);

    int n,ds,hs,ws;
    int cell_depth = octree_ind_to_dense_ind(&grad_in2, in2_grid_idx, in2_bit_idx, &n, &ds,&hs,&ws);
    int cell_width = width_from_depth(cell_depth);
    int cell_width3 = cell_width * cell_width * cell_width;
            
    for(int f = 0; f < grad_in2.feature_size; ++f) {
      grad_in2.data[leaf_idx * grad_in2.feature_size + f] = 0;
    }

    for(int d = ds; d < (ds+cell_width); ++d) {
      for(int h = hs; h < (hs+cell_width); ++h) {
        for(int w = ws; w < (ws+cell_width); ++w) {
          int gd = d / 8;
          int gh = h / 8;
          int gw = w / 8;
          int bd = d % 8;
          int bh = h % 8;
          int bw = w % 8;

          int out_grid_idx = octree_grid_idx(&grad_out, n, gd,gh,gw);
          const ot_tree_t* out_tree = octree_get_tree(&grad_out, out_grid_idx);
          int out_bit_idx = tree_bit_idx(out_tree, bd,bh,bw);
          ot_data_t* out_data = octree_get_data(&grad_out, out_grid_idx);
          int out_data_idx = tree_data_idx(out_tree, out_bit_idx, grad_out.feature_size);

          // int out_cell_width = width_from_bit_idx(out_bit_idx);
          // int out_cell_width3 = out_cell_width*out_cell_width*out_cell_width;
          
          for(int f = 0; f < grad_in2.feature_size; ++f) {
            float val = out_data[out_data_idx + in1_feature_size + f];
            grad_in2.data[leaf_idx * grad_in2.feature_size + f] += val / cell_width3;
          }
        }
      }
    }
  }
}

void octree_concat_ds_bwd_gpu(const octree* in1, const octree* in2, const octree* grad_out, bool do_grad_in2, octree* grad_in1, octree* grad_in2) {
  if(DEBUG) { printf("[DEBUG] octree_concat_bwd_gpu\n"); }

  octree_resize_as_gpu(in1, grad_in1);
  octree_cpy_trees_gpu_gpu(in1, grad_in1);
  octree_cpy_prefix_leafs_gpu_gpu(in1, grad_in1);

  kernel_concat_ds_bwd1<<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
     *grad_in1, in1->n_leafs, *grad_out
  );
  CUDA_POST_KERNEL_CHECK;

  if(do_grad_in2) {
    octree_resize_as_gpu(in2, grad_in2);
    octree_cpy_trees_gpu_gpu(in2, grad_in2);
    octree_cpy_prefix_leafs_gpu_gpu(in2, grad_in2);

    kernel_concat_ds_bwd2<<<GET_BLOCKS(in2->n_leafs), CUDA_NUM_THREADS>>>(
       *grad_in2, in2->n_leafs, *grad_out, in1->feature_size
    );
    CUDA_POST_KERNEL_CHECK;
  }
}




__global__ void kernel_concat_dense(ot_data_t* out, int n_leafs, const octree in1, const ot_data_t* in2, const ot_size_t feature_size1, const ot_size_t feature_size2, const ot_size_t feature_size_out) {
  const int dense_depth = 8 * in1.grid_depth;
  const int dense_height = 8 * in1.grid_height;
  const int dense_width = 8 * in1.grid_width;
  
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    octree_cpy_leaf(in1.data + leaf_idx * feature_size1, feature_size1, out + leaf_idx * feature_size_out);

    int grid_idx = leaf_idx_to_grid_idx(&in1, leaf_idx);
    const ot_tree_t* tree = octree_get_tree(&in1, grid_idx);

    int cum_n_leafs = in1.prefix_leafs[grid_idx];
    int data_idx = leaf_idx - cum_n_leafs;
    int bit_idx = data_idx_to_bit_idx(tree, data_idx);

    int n,ds,hs,ws;
    int depth = octree_ind_to_dense_ind(&in1, grid_idx, bit_idx, &n, &ds,&hs,&ws);
    int width = width_from_depth(depth);

    for(int f = 0; f < feature_size2; ++f) {
      ot_data_t val = 0;
      for(int d = ds; d < ds+width; ++d) {
      for(int h = hs; h < hs+width; ++h) {
      for(int w = ws; w < ws+width; ++w) {
        int dense_idx = (((n * feature_size2 + f) * dense_depth + d) * dense_height + h) * dense_width + w;
        float add = in2[dense_idx];
        // float add = in2[dense_idx] / (width*width*width);
        val += add;
      }
      }
      }

      out[leaf_idx * feature_size_out + feature_size1 + f] = val / (width*width*width);
      // out[leaf_idx * feature_size_out + feature_size1 + f] = val;
    }
  }
}

void octree_concat_dense_gpu(const octree* in1, const ot_data_t* in2, ot_size_t feature_size2, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_concat_dense_gpu\n"); }

  ot_size_t feature_size1 = in1->feature_size;
  ot_size_t feature_size_out = feature_size1 + feature_size2;

  octree_resize_gpu(in1->n, in1->grid_depth, in1->grid_height, in1->grid_width, feature_size_out, in1->n_leafs, out);
  octree_cpy_trees_gpu_gpu(in1, out);
  octree_cpy_prefix_leafs_gpu_gpu(in1, out);

  kernel_concat_dense<<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
      out->data, in1->n_leafs, *in1, in2, feature_size1, feature_size2, feature_size_out
  );
  CUDA_POST_KERNEL_CHECK;
}


template <bool do_grad_in2>
__global__ void kernel_concat_dense_bwd(ot_data_t* grad_in1, ot_data_t* grad_in2, int n_leafs, const octree grad_out, const ot_size_t feature_size1, const ot_size_t feature_size2, const ot_size_t feature_size_out) {
  const int dense_depth = 8 * grad_out.grid_depth;
  const int dense_height = 8 * grad_out.grid_height;
  const int dense_width = 8 * grad_out.grid_width;

  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    octree_cpy_leaf(grad_out.data + leaf_idx * feature_size_out, feature_size1, grad_in1 + leaf_idx * feature_size1);

    if(do_grad_in2) {
      int grid_idx = leaf_idx_to_grid_idx(&grad_out, leaf_idx);
      const ot_tree_t* tree = octree_get_tree(&grad_out, grid_idx);

      int cum_n_leafs = grad_out.prefix_leafs[grid_idx];
      int data_idx = leaf_idx - cum_n_leafs;
      int bit_idx = data_idx_to_bit_idx(tree, data_idx);

      int n,ds,hs,ws;
      int depth = octree_ind_to_dense_ind(&grad_out, grid_idx, bit_idx, &n, &ds,&hs,&ws);
      int width = width_from_depth(depth);

      for(int f = 0; f < feature_size2; ++f) {
        ot_data_t val = grad_out.data[leaf_idx * grad_out.feature_size + feature_size1 + f];
        for(int d = ds; d < ds+width; ++d) {
        for(int h = hs; h < hs+width; ++h) {
        for(int w = ws; w < ws+width; ++w) {
          grad_in2[(((n * feature_size2 + f) * dense_depth + d) * dense_height + h) * dense_width + w] = val;
        }
        }
        }
      }
    }
  }
}

void octree_concat_dense_bwd_gpu(const octree* in1, const ot_data_t* in2, ot_size_t feature_size2, const octree* grad_out, bool do_grad_in2, octree* grad_in1, ot_data_t* grad_in2) {
  if(DEBUG) { printf("[DEBUG] octree_concat_dense_bwd_gpu\n"); }

  octree_resize_as_gpu(in1, grad_in1);
  octree_cpy_trees_gpu_gpu(in1, grad_in1);
  octree_cpy_prefix_leafs_gpu_gpu(in1, grad_in1);
  
  ot_size_t feature_size1 = in1->feature_size;
  ot_size_t feature_size_out = feature_size1 + feature_size2;

  if(do_grad_in2) {
    kernel_concat_dense_bwd<true><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2, in1->n_leafs, *grad_out, feature_size1, feature_size2, feature_size_out
    );
  }
  else {
    kernel_concat_dense_bwd<false><<<GET_BLOCKS(in1->n_leafs), CUDA_NUM_THREADS>>>(
       grad_in1->data, grad_in2, in1->n_leafs, *grad_out, feature_size1, feature_size2, feature_size_out
    );

  }
  CUDA_POST_KERNEL_CHECK;
}
