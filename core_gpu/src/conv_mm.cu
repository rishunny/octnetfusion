#include "hip/hip_runtime.h"
// Copyright (c) 2017, The OctNet authors
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL OCTNET AUTHORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "octnet/gpu/conv.h"
#include "octnet/gpu/gpu.h"
#include "octnet/gpu/oc2col.h"
#include "octnet/gpu/col2oc.h"
#include "octnet/gpu/buffer.h"

#include <thrust/fill.h>
#include <thrust/execution_policy.h>


void print_matrix_gpu(const ot_data_t* data_d, int rows, int cols) {
  ot_data_t* data_h = device_to_host_malloc(data_d, rows*cols);
  int idx = 0;
  printf("[");
  for(int row = 0; row < rows; ++row) {
    if(row > 0) printf(" ");
    printf("[ ");
    for(int col = 0; col < cols; ++col) {
      printf("%f", data_h[idx]);
      idx++;
      if(col < cols-1) {
        printf(", ");
      }
    }
    if(row < rows - 1) {
      printf(" ], \n");
    }
    else {
      printf(" ]] \n");
    }
  }
  delete[] data_h;
}


__global__ void kernel_conv_mm_add_bias(ot_data_t* out, int n_leafs, int channels_out, const ot_data_t* bias) {
  CUDA_KERNEL_LOOP(leaf_idx, n_leafs) {
    for(int f = 0; f < channels_out; ++f) {
      out[leaf_idx * channels_out + f] += bias[f];
    }
  }
}

void octree_conv_mm_block_gpu(hipblasHandle_t cublas_handle, const octree* in, const ot_data_t* weights, int leafs_offset, int n_leafs, octree* out) {
  ot_data_t_buffer_gpu& col_buffer = ot_data_t_buffer_gpu::i();
  col_buffer.resize(long(n_leafs) * K333 * in->feature_size);

  oc2col_gpu(in, col_buffer.data(), col_buffer.capacity(), leafs_offset, n_leafs);

  float alpha = 1;
  float beta = 0;
  int m = out->feature_size;
  int n = n_leafs;
  int k = in->feature_size * K333;
  CUBLAS_CHECK(
    hipblasSgemm(
      cublas_handle,
      HIPBLAS_OP_T, HIPBLAS_OP_N,
      m, n, k,
      &alpha,
      weights, k,
      col_buffer.data(), k,
      &beta,
      out->data + leafs_offset * out->feature_size, m
    )
  );
  
  // printf("weights=\n");
  // print_matrix_gpu(weights, out->feature_size, K333*in->feature_size);
  // printf("out=\n");
  // print_matrix_gpu(out->data, out->n_leafs, out->feature_size);
  // printf("col_buffer=\n");
  // print_matrix_gpu(col_buffer.data(), n_leafs, K333*in->feature_size);
}

void octree_conv_mm_gpu(hipblasHandle_t cublas_handle, const octree* in, const ot_data_t* weights, const ot_data_t* bias, int channels_out, int n_grids, octree* out) {
  if(DEBUG) { printf("[DEBUG] octree_conv_mm_gpu\n"); }

  octree_resize_gpu(in->n, in->grid_depth, in->grid_height, in->grid_width, channels_out, in->n_leafs, out);
  octree_cpy_scalars(in, out);
  out->feature_size = channels_out;
  octree_cpy_trees_gpu_gpu(in, out);
  octree_cpy_prefix_leafs_gpu_gpu(in, out);

  if(n_grids < 0) {
    int leafs_offset = 0;
    int n_leafs = in->n_leafs;
    octree_conv_mm_block_gpu(cublas_handle, in, weights, leafs_offset, n_leafs, out);
  }
  else {
    if(n_grids == 0) {
      n_grids = in->grid_depth * in->grid_height * in->grid_width;
    }
    int n_blocks = octree_num_blocks(in);
    ot_size_t* prefix_leafs_cpu = device_to_host_malloc<ot_size_t>(in->prefix_leafs, n_blocks);
    int grid_idx = 0;
    while(grid_idx < n_blocks) {
      int leafs_offset = prefix_leafs_cpu[grid_idx]; 
      int n_leafs;
      if(grid_idx + n_grids < n_blocks) {
        n_leafs = prefix_leafs_cpu[grid_idx + n_grids] -  prefix_leafs_cpu[grid_idx];
      }
      else {
        n_leafs = in->n_leafs -  prefix_leafs_cpu[grid_idx];
      }
      octree_conv_mm_block_gpu(cublas_handle, in, weights, leafs_offset, n_leafs, out);
      grid_idx += n_grids;
    }
    delete[] prefix_leafs_cpu;
  }

  // add bias
  kernel_conv_mm_add_bias<<<GET_BLOCKS(out->n_leafs), CUDA_NUM_THREADS>>>(
     out->data, out->n_leafs, out->feature_size, bias
  );
  CUDA_POST_KERNEL_CHECK;
}

void octree_conv_mm_bwd_block_gpu(hipblasHandle_t cublas_handle, const octree* grad_out, const ot_data_t* weights, int leafs_offset, int n_leafs, bool atomic, octree* grad_in) {
  ot_data_t_buffer_gpu& col_buffer = ot_data_t_buffer_gpu::i();
  col_buffer.resize(long(n_leafs) * K333 * grad_in->feature_size);

  float alpha = 1;
  float beta = 0;
  int m = grad_in->feature_size * K333;
  int n = n_leafs;
  int k = grad_out->feature_size;
  CUBLAS_CHECK(
    hipblasSgemm(
      cublas_handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      m, n, k,
      &alpha,
      weights, m,
      grad_out->data + leafs_offset * grad_out->feature_size, k,
      &beta,
      col_buffer.data(), m
    )
  );

  if(atomic) {
    col2oc_atomic_gpu(col_buffer.data(), grad_in, leafs_offset, n_leafs);
  }
  else {
    col2oc_gpu(col_buffer.data(), grad_in, leafs_offset, n_leafs);
  }
  

  // printf("weights=\n");
  // print_matrix_gpu(weights, grad_out->feature_size, K333*grad_in->feature_size);
  // printf("grad_out=\n");
  // print_matrix_gpu(grad_out->data, grad_in->n_leafs, grad_out->feature_size);
  // printf("col_buffer=\n");
  // print_matrix_gpu(col_buffer.data(), n_leafs, K333*grad_in->feature_size);
}

void octree_conv_mm_bwd_gpu(hipblasHandle_t cublas_handle, const octree* grad_out, const ot_data_t* weights, int channels_in, int n_grids, octree* grad_in) {
  if(DEBUG) { printf("[DEBUG] octree_conv_mm_bwd_gpu\n"); }

  octree_resize_gpu(grad_out->n, grad_out->grid_depth, grad_out->grid_height, grad_out->grid_width, channels_in, grad_out->n_leafs, grad_in);
  octree_cpy_scalars(grad_out, grad_in);
  grad_in->feature_size = channels_in;
  octree_cpy_trees_gpu_gpu(grad_out, grad_in);
  octree_cpy_prefix_leafs_gpu_gpu(grad_out, grad_in);

  bool atomic = false;
  if(n_grids < 0) {
    int leafs_offset = 0;
    int n_leafs = grad_out->n_leafs;
    octree_conv_mm_bwd_block_gpu(cublas_handle, grad_out, weights, leafs_offset, n_leafs, atomic, grad_in);
  }
  else {
    if(n_grids == 0) {
      n_grids = grad_out->grid_depth * grad_out->grid_height * grad_out->grid_width;
    }
    else {
      atomic = true;
      octree_fill_data_gpu(grad_in, 0);
    }
    int n_blocks = octree_num_blocks(grad_out);
    ot_size_t* prefix_leafs_cpu = device_to_host_malloc<ot_size_t>(grad_out->prefix_leafs, n_blocks);
    int grid_idx = 0;
    while(grid_idx < n_blocks) {
      int leafs_offset = prefix_leafs_cpu[grid_idx]; 
      int n_leafs;
      if(grid_idx + n_grids < n_blocks) {
        n_leafs = prefix_leafs_cpu[grid_idx + n_grids] -  prefix_leafs_cpu[grid_idx];
      }
      else {
        n_leafs = grad_out->n_leafs -  prefix_leafs_cpu[grid_idx];
      }
      octree_conv_mm_bwd_block_gpu(cublas_handle, grad_out, weights, leafs_offset, n_leafs, atomic, grad_in);
      grid_idx += n_grids;
    }
    delete[] prefix_leafs_cpu;
  }
  
  // printf("grad_in=\n");
  // print_matrix_gpu(grad_in->data, grad_in->n_leafs, grad_in->feature_size);
}



void octree_conv_mm_wbwd_block_gpu(hipblasHandle_t cublas_handle, const octree* in, const octree* grad_out, const float scale, int leafs_offset, int n_leafs, ot_data_t* grad_weights, ot_data_t* grad_bias) {

  ot_data_t_buffer_gpu& col_buffer = ot_data_t_buffer_gpu::i();
  col_buffer.resize(long(n_leafs) * K333 * in->feature_size);

  oc2col_gpu(in, col_buffer.data(), col_buffer.capacity(), leafs_offset, n_leafs);

  float alpha = scale;
  float beta = 1;
  int m = in->feature_size * K333;
  int n = grad_out->feature_size;
  int k = n_leafs;
  CUBLAS_CHECK(
    hipblasSgemm(
      cublas_handle,
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      m, n, k,
      &alpha,
      col_buffer.data(), m,
      grad_out->data + leafs_offset * grad_out->feature_size, n,
      &beta,
      grad_weights, m
    )
  );

  thrust::fill(thrust::device, col_buffer.data(), col_buffer.data() + n_leafs, 1.f);

  alpha = scale;
  beta = 1;
  m = grad_out->feature_size;
  n = n_leafs;
  CUBLAS_CHECK(
    hipblasSgemv(
      cublas_handle,
      HIPBLAS_OP_N,
      m, n,
      &alpha,
      grad_out->data + leafs_offset * grad_out->feature_size, m,
      col_buffer.data(), 1,
      &beta,
      grad_bias, 1
    )
  );
}

void octree_conv_mm_wbwd_gpu(hipblasHandle_t cublas_handle, const octree* in, const octree* grad_out, const float scale, int n_grids, ot_data_t* grad_weights, ot_data_t* grad_bias) {
  if(DEBUG) { printf("[DEBUG] octree_conv_mm_wbwd_gpu\n"); }

  if(n_grids < 0) {
    int leafs_offset = 0;
    int n_leafs = in->n_leafs;
    octree_conv_mm_wbwd_block_gpu(cublas_handle, in, grad_out, scale, leafs_offset, n_leafs, grad_weights, grad_bias);
  }
  else {
    if(n_grids == 0) {
      n_grids = in->grid_depth * in->grid_height * in->grid_width;
    }
    int n_blocks = octree_num_blocks(in);
    ot_size_t* prefix_leafs_cpu = device_to_host_malloc<ot_size_t>(in->prefix_leafs, n_blocks);
    int grid_idx = 0;
    while(grid_idx < n_blocks) {
      int leafs_offset = prefix_leafs_cpu[grid_idx]; 
      int n_leafs;
      if(grid_idx + n_grids < n_blocks) {
        n_leafs = prefix_leafs_cpu[grid_idx + n_grids] -  prefix_leafs_cpu[grid_idx];
      }
      else {
        n_leafs = in->n_leafs -  prefix_leafs_cpu[grid_idx];
      }
      octree_conv_mm_wbwd_block_gpu(cublas_handle, in, grad_out, scale, leafs_offset, n_leafs, grad_weights, grad_bias);
      grid_idx += n_grids;
    }
    delete[] prefix_leafs_cpu;
  }
}
